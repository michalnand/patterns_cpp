#include "cuda_common.cuh"

CudaCommon::CudaCommon()
{

}

CudaCommon::~CudaCommon()
{

}


CudaCommon& CudaCommon::get_instance()
{
  static CudaCommon instance;
  return instance;
}

unsigned int CudaCommon::get_device_count()
{
  int result = 0;
  hipGetDeviceCount(&result);
  return result;
}

void CudaCommon::set_device(unsigned int device_idx)
{
  hipSetDevice(device_idx);
}

unsigned int CudaCommon::get_threads_per_block(unsigned int size)
{
  if (size <= 256)
    return 32;
  else
    return 256;
}

unsigned int CudaCommon::get_number_of_blocks(unsigned int size)
{
  unsigned int tmp = get_threads_per_block(size);
  return (size + tmp -1)/tmp;
}
