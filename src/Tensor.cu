#include "Tensor.h"

Tensor::Tensor()
{
  this->w     = 0;
  this->h     = 0;
  this->d     = 0;
  this->size  = 0;

  this->v     = nullptr;
}

Tensor::Tensor(unsigned int d)
{
  this->w     = 0;
  this->h     = 0;
  this->d     = 0;
  this->size  = 0;

  this->v     = nullptr;

  init(1, 1, d);
}

Tensor::Tensor(unsigned int w, unsigned int d, unsigned int h)
{
  this->w     = 0;
  this->h     = 0;
  this->d     = 0;
  this->size  = 0;

  this->v     = nullptr;

  init(w, h, d);
}

Tensor::Tensor(sGeometry geometry)
{
  this->w     = 0;
  this->h     = 0;
  this->d     = 0;
  this->size  = 0;

  this->v     = nullptr;

  init(geometry.w, geometry.h, geometry.d);
}

Tensor::Tensor(Tensor &rhs)
{
  this->w     = 0;
  this->h     = 0;
  this->d     = 0;
  this->size  = 0;

  this->v     = nullptr;

  init(rhs.w, rhs.h, rhs.d);
}

Tensor::Tensor(const Tensor &rhs)
{
  this->w     = 0;
  this->h     = 0;
  this->d     = 0;
  this->size  = 0;

  this->v     = nullptr;
}

Tensor::~Tensor()
{
  uninit();
}


void Tensor::init(unsigned int d)
{
  uninit();
  init(1, 1, d);
}

void Tensor::init(unsigned int w, unsigned int h, unsigned int d)
{
  uninit();

  this->w = w;
  this->h = h;
  this->d = d;

  this->size = w*h*d;

  #ifdef _USE_CUDA_
    hipMalloc(&v, size*sizeof(float));
  #else

  #endif

}

void Tensor::init(sGeometry geometry)
{
  uninit();
  init(geometry.w, geometry.h, geometry.d);
}

void Tensor::uninit()
{
  if (v != nullptr)
  {
    #ifdef _USE_CUDA_
      hipFree(v);
    #else
      delete v;
    #endif

    w     = 0;
    h     = 0;
    d     = 0;
    size  = 0;

    v     = nullptr;
  }
}

void Tensor::clear()
{

}


Tensor& Tensor::operator=(Tensor &rhs)
{
  return *this;
}

Tensor& Tensor::operator=(float *host_src)
{
  return *this;
}

Tensor& Tensor::operator=(std::vector<float> &host_src)
{
  return *this;
}
